#include "hip/hip_runtime.h"
#include <partition.h>

__global__ void partition(int* A, int na, int* B, int nb, int* C){

	int nbThreads = blockDim.x * gridDim.x;		// number of threads
	int tid = blockIdx.x*blockDim.x+threadIdx.x;	// thread ID
	int load = (na+nb)/nbThreads;			// size of each subarray
	int index = tid*load;				// starting index in C

	// search zone: indices of the top-right cell of the central diagonal
	int a_top = (index>na)? na:index;		// col index (in A)
	int b_top = (index>na)? index-na:0;		// row index (in B)
	int a_bot = b_top;				// top left col index

	// binary search (dichotomy)
	int a, b, offset, aid, bid;
	if(tid ==0) {
		// thread 0 always starts at (0,0)
		aid = 0;	bid = 0;
	}
	else {
		while(true) {
			// get mid cell of the (sub-)diagonal
			offset = (a_top - a_bot) / 2;
			a = a_top - offset;		b = b_top + offset;

			// check if point found
			if(A[a]>B[b-1]){
				if(A[a-1]<=B[b]){
					// point found
					aid = a;	bid = b;	break;
				}
				else{
					// restrict search to lower half
					a_top = a-1;	b_top = b+1;
				}
			}
			else{
				// restrict search to upper half
				a_bot = a+1;
			}
		}
	}

	printf("[%d] (%d,%d); %d\n", tid, aid, bid, index);
	merge(A, na, aid, B, nb, bid, C, index, load);
}


__global__ void partition2(int* A, int na, int* B, int nb, int* C) {

	int nbThreads = blockDim.x * gridDim.x;			// number of threads
	int tid = blockIdx.x*blockDim.x+threadIdx.x;	// thread ID
	int load = (na+nb)/nbThreads;					// size of each subarray
	int index = tid*load;							// starting index in C

	// search zone: indices of the top-right cell of the central diagonal
	int a_top = (index>na)? na:index;		// col index (in A)
	int b_top = (index>na)? index-na:0;		// row index (in B)
	int a_bot = b_top;						// top left col index

	// binary search (dichotomy)
	int a, b, offset, aid, bid;
	if(tid ==0) {
		// thread 0 always starts at (0,0)
		aid = 0;	bid = 0;
	}
	else {
		while(true) {
			// get mid cell of the (sub-)diagonal
			offset = (a_top - a_bot) / 2;
			a = a_top - offset;		b = b_top + offset;

			// check if point found
			if(A[a]>B[b-1]){
				if(A[a-1]<=B[b]){
					// point found
					aid = a;	bid = b;	break;
				}
				else{
					// restrict search to lower half
					a_top = a-1;	b_top = b+1;
				}
			}
			else{
				// restrict search to upper half
				a_bot = a+1;
			}
		}
	}

	printf("[%d] (%d,%d); %d\n", tid, aid, bid, index);

	if(load<1024)	merge2<<<1,load>>>(A, na, aid, B, nb, bid, C, index, load);
	else		merge2<<<1,1024>>>(A, na, aid, B, nb, bid, C, index, load);
}
