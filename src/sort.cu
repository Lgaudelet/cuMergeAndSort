#include "hip/hip_runtime.h"
#include <sort.h>

__host__ __device__ void bubbleSort(int* array, int size) {
      bool swapped = true;
      int j = 0;
      int tmp;
      while(swapped) {
            swapped = false;
            j++;
            for(int i=0; i<size-j; i++) {
                  if(array[i] > array[i+1]) {
                        tmp = array[i];
                        array[i] = array[i+1];
                        array[i+1] = tmp;
                        swapped = true;
                  }
            }
      }
}

__global__ void initial_sort(int* array, int size, int grain_size) {

	int tid = blockIdx.x*blockDim.x + threadIdx.x; // thread ID
	int index = tid*grain_size;

	while(index < size) {
		int n = (index+grain_size>size)? grain_size-index+size:grain_size;
		bubbleSort(array+index,n);
		index+=gridDim.x*blockDim.x*grain_size;
	}
}
