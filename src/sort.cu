#include "hip/hip_runtime.h"
#include <sort.h>

__host__ __device__ void bubbleSort(int* array, int size) {
      bool swapped = true;
      int j = 0;
      int tmp;
      while(swapped) {
            swapped = false;
            j++;
            for(int i=0; i<size-j; i++) {
                  if(array[i] > array[i+1]) {
                        tmp = array[i];
                        array[i] = array[i+1];
                        array[i+1] = tmp;
                        swapped = true;
                  }
            }
      }
}

__global__ void initial_sort(int* array, int size, int grain_size) {

	int tid = blockIdx.x*blockDim.x + threadIdx.x; // thread ID
	int index = tid*grain_size;

	while(index < size) {
		int n = (index+grain_size>size)? grain_size-index+size:grain_size;
		bubbleSort(array+index,n);
		index+=gridDim.x*blockDim.x*grain_size;
	}
}

__global__ void parallel_merge(int* input_array, int size, int* output_array, int subarray_size, int part_size) {
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int nPartitions = ceil((float)subarray_size/part_size);

	int shift_A = 2*tid*subarray_size;
	int shift_B = 2*(tid+1)*subarray_size;

	int na = subarray_size;
	int nb = (shift_B+subarray_size>size)? size-shift_B:subarray_size;

	partition2<<<1,nPartitions>>>(input_array+shift_A, na, input_array+shift_B, nb, output_array+shift_A); 

}


void msWrapper(int* input_array, int size, int* output_array, int grain_exp) {

	int p = nextpow2(size) - grain_exp;
	if(p<0) {
		std::cout << "input array is too small for specified grain" << std::endl;
		exit(-1);
	}
	
	int *tmp, *tmp2;
	int subarray_size = std::pow(2,grain_exp);

	// initial sorting of the array
	hipMalloc(&tmp, size*sizeof(int));
	hipMemcpy(tmp, input_array, size*sizeof(int), hipMemcpyHostToDevice);

std::cout << "initial_sort:" << subarray_size << std::endl;
	initial_sort<<<1, std::ceil((float)size/subarray_size)>>>(tmp, size, subarray_size);	

	// merging arrays two by two until complete sorting
	p = std::pow(2,p);
	while(p>1) {
		hipMalloc(&tmp2, size*sizeof(int));

std::cout << (p>>1) << " x " << subarray_size << std::endl;
		parallel_merge <<<1,(p>>1)>>> (tmp, size, tmp2, subarray_size);
	
		hipFree(tmp);
		tmp = tmp2;

		subarray_size<<=1;	
		p >>= 1; //divides p by 2
	}
	
	hipMemcpy(output_array, tmp, size*sizeof(int), hipMemcpyDeviceToHost);

}

