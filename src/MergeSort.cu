#include "hip/hip_runtime.h"
#include <MergeSort.h>

//template <typename type>
//__host__ __device__ void merge(type* A, int aid, type* B, int bid, type* C, int cid, int load) {
__host__ __device__ void merge(int* A, int aid, int* B, int bid, int* C, int cid, int load) {
	for(int t=0; t<load; t++) {
		if(A[aid] < B[bid]) {
			C[cid-t] = A[aid];	aid++;
		}
		else{
			C[cid-t] = B[bid];	bid++;
		}
	}
}

//template <typename type>
//__global__ void partitionning(type* A, int na, type* B, int nb, type* C, int* Adiag, int* Bdiag){
__global__ void partitionning(int* A, int na, int* B, int nb, int* C, int* Adiag, int* Bdiag){

	printf("in\n");

	int nbThreads = blockDim.x * gridDim.x;
	
	int tid = blockIdx.x*blockDim.x+threadIdx.x;	// thread ID
	int load = (na+nb)/nbThreads;			// the size of each thread's sub-array
	int index = tid*load;				// starting index in C
	
	if(tid==0) {
		Bdiag[nbThreads-1] = nb; 
		Adiag[nbThreads-1] = na;
	}

	printf("[%d] in\n", tid);

	// col index (in A) and row index (in B) of the top-right cell of the central diag
	int a_top = (index>na)? na:index;		
	int b_top = (index>na)? na-index:0;	
	int a_bot = b_top;	// top left col index

	// binary search
	int a,b,offset;
	while(true){
		offset = (a_top - a_bot) / 2;
		a = a_top - offset;
		b = b_top + offset;

		if(A[a]>B[b-1]){
			if(A[a-1]<=B[b]){
				Adiag[tid] = a;
				Bdiag[tid] = b;
				break;
			}
			else{
				a_top = a-1;
				b_top = a+1;
			}
		}
		else{
			a_bot = a+1;
		}

	}

	printf("[%d] merge\n", tid);
	merge(A, Adiag[tid], B, Bdiag[tid], C, index, load);
	printf("[%d] out\n", tid);

}


