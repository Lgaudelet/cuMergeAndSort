#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <chCommandLine.h>
#include <chTimer.hpp>
#include <hip/hip_runtime.h>
#include <MergeSort.h>

const static int DEFAULT_N = 100;

// function prototypes
template <typename type> void init_tab(type* array, int n);
template <typename type> void cpu_tri(type* array, int n);
template <typename type> void print_tab(type* array, int n);
template <typename type> bool is_sorted(type* array, int n);

void print_help( char* argv);

//main
int main(int argc, char* argv[]){
	
	// print help
	bool      help = chCommandLineGetBool("h", argc, argv);
	if(!help) help = chCommandLineGetBool("help", argc, argv);
	if(help)  {
		print_help(argv[0]); return 0;
	}

	// size
	int n = -1;
	chCommandLineGet<int>(&n, "n", argc, argv);
	chCommandLineGet<int>(&n, "size", argc, argv);
	n = (n!=-1)? n:DEFAULT_N;

	// allouer mémoire et initialiser

	std::cout << "Initialisation...\t" << std::flush;
		// cpu
	int* cpu_v = (int*)malloc(n*sizeof(int));
	int* out = (int*)malloc(n*sizeof(int));
	init_tab(cpu_v, n);

		// gpu
	int na, nb;
	na = floor(n/2);
	nb = ceil(n/2);
	//std::cout << na << " " << nb << std::endl;

	int *A, *B, *C;
	hipMalloc(&A, na*sizeof(int));
	hipMalloc(&B, nb*sizeof(int));
	hipMalloc(&C, n*sizeof(int));

	cpu_tri(cpu_v, na);
	cpu_tri(cpu_v+na, nb);	
	hipMemcpy(A, cpu_v, na*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B, cpu_v+na, nb*sizeof(int), hipMemcpyHostToDevice);

	//print_tab(cpu_v, n);

	// trier et chronométrer
	int gridSize = 1;
	int blockSize = 4;

	int *Adiag, *Bdiag;
	hipMalloc(&Adiag, gridSize*blockSize*sizeof(int));
	hipMalloc(&Bdiag, gridSize*blockSize*sizeof(int));

	if(!A || !B || !C || !Adiag || !Bdiag) {
		std::cout << "memory alloc error" << std::endl;
		return -1;
	}
	std::cout << "done" << std::endl;

	std::cout << "Partitionning...\t" << std::flush;
	//partitionning<int><<<gridSize, blockSize>>>(A, na, B, nb, C, Adiag, Bdiag);
	partitionning<<<gridSize, blockSize>>>(A, na, B, nb, C, Adiag, Bdiag);
	hipDeviceSynchronize();
	std::cout << "done" << std::endl;

	// D2H
	std::cout << "transfert D2H...\t" << std::flush;
	hipMemcpy(out, C, n*sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "done" << std::endl;

	// afficher
	print_tab(out, n);
	std::cout << is_sorted(out, n) << std::endl;

	// return 0
	return 0;
}

template <typename type>
void init_tab(type* array, int n) {
	// génère n entiers aléatoirements dans array
	for(int i=0;i<n;i++){
		array[i]=rand()%10;
	}
}

template <typename type>
void cpu_tri(type* array, int n) {
	bool swap=true;
	for(int i=0;(i<n)&&swap;i++){
		swap = false;
		for(int j=0;(j<n-i-1);j++){
			if(array[j]>array[j+1]){
				int tmp = array[j];
				array[j] = array[j+1];
				array[j+1] = tmp;
				swap = true;
			}
		}
	}
}

template <typename type>
bool is_sorted(type* array, int n) {
	for(int i=0; i<n; i++) {
		if(array[i]>array[i+1]) return false;
	}
	return true;
}


template <typename type>
void print_tab(type* array, int n) {
	for(int i=0; i<n; i++) {
		std::cout << array[i] << " ";
	}
	std::cout << std::endl;
}

void print_help( char* argv) {
/*
	cout	<< "Help:" << endl
		<< "  Usage: " << endl
		<< "  " << argv << " [options][-n <repressor-number> ]" << endl
		<< endl
		<< "  -n|--repressor-number" << endl
		<< "      number of repressors to be used, must be an odd integer" << endl
		<< endl;
*/
}
