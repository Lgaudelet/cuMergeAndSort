#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <chTimer.hpp>
#include <chCommandLine.h>

#include <utils.h>
#include <merge.h>
#include <partition.h>
#include <sort.h>


const static int DEFAULT_N = 100;
const static int DEFAULT_PARTSIZE = 32;

void print_help( char* argv);

//main
int main(int argc, char* argv[]){
	
	// print help
	bool      help = chCommandLineGetBool("h", argc, argv);
	if(!help) help = chCommandLineGetBool("help", argc, argv);
	if(help)  {
		print_help(argv[0]); return 0;
	}

	// size
	int n = -1;
	chCommandLineGet<int>(&n, "n", argc, argv);
	chCommandLineGet<int>(&n, "size", argc, argv);
	n = (n!=-1)? n:DEFAULT_N;

	// size of partition
	int partSize = -1;
	chCommandLineGet<int>(&partSize, "p", argc, argv);
	chCommandLineGet<int>(&partSize, "partSize", argc, argv);
	partSize = (partSize!=-1)? partSize:DEFAULT_PARTSIZE;

	// memory allocation and initialisation
	std::cout << "Initialization...\t" << std::flush;
	int* cpu_v = (int*)malloc(n*sizeof(int));
	int* out = (int*)malloc(n*sizeof(int));
	init_array(cpu_v, n);
	std::cout << "done" << std::endl;

	// gpu sort
	std::cout << "gpu sort...\t\t" << std::flush;
	ChTimer kernel;
	kernel.start();
	msWrapper(cpu_v, n, out, partSize);
	kernel.stop();
	std::cout << "done" << std::endl;


	// compare results
	ChTimer cpuTimer;
	bool compare_cpu = (chCommandLineGetBool("c", argc, argv))?
		true:chCommandLineGetBool("compare-cpu", argc, argv);
	if(compare_cpu) {
		std::cout << "cpu sort...\t" << std::flush;
		cpuTimer.start();
		bubbleSort(cpu_v, n);
		cpuTimer.stop();
		std::cout << "done" << std::endl << std::endl;
	}

	// display performances
	std::cout << "Results...\t" << std::flush;
	std::cout << "\tsorted=" << is_sorted(out, n) << std::endl;
	std::cout << "\tgpu time: " << 1e3*kernel.getTime() << "ms" << std::endl;
	if(compare_cpu)
		std::cout <<"\tcpu time: "<<1e3*cpuTimer.getTime()<<"ms"<<std::endl;

	// application thats's very nice and all
	/*float a = 0;
	std::cout << "Entrez un pourcentage * souhaité pour connaître l'année de panne correspondant: " <<std::endl;
	std::cin >> a;

	int result = floor(a/100*n);

	std::cout << std::endl;
	std::cout << "Il y a " << a << "% des appareils qui tombent en panne avant " << out[result]<< " ans" << std::endl;
	std::cout << std::endl;
	std::cout << std::endl;
	std::cout << "Informations complémentaires:" << std::endl;
	std::cout << "25% des appareils tombent en panne avant : \t" << out[n/4] <<" ans"<< std::endl;
	std::cout << "la durée de vie médiane des appareils est : \t" << out[n/2] <<" ans"<< std::endl;
	std::cout << "25% des appareils tombent en panne après : \t" << out[3*n/4] << " ans" << std::endl;
	*/

	//free
	free(cpu_v);	free(out);

	return 0;
}

void print_help( char* argv) {

	std::cout << "Help:" << std::endl
		<< "  Usage: " << std::endl
		<< "  " << argv << " [options][-n <size>][-b <blockSize>]" << std::endl
		<< std::endl
		<< "  -n|--size" << std::endl
		<< "      size of input array" << std::endl
		<< "  -p|--partSize" << std::endl
		<< "      size of each partition" << std::endl
		<< std::endl;

}
