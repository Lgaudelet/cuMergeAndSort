#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <chTimer.hpp>
#include <chCommandLine.h>

#include <utils.h>
#include <merge.h>
#include <partition.h>
#include <sort.h>

const static int DEFAULT_N = 12;
const static int DEFAULT_BLOCKSIZE = 4;

void print_help( char* argv);

//main
int main(int argc, char* argv[]){
	
	// print help
	bool      help = chCommandLineGetBool("h", argc, argv);
	if(!help) help = chCommandLineGetBool("help", argc, argv);
	if(help)  {
		print_help(argv[0]); return 0;
	}

	// size
	int n = -1;
	chCommandLineGet<int>(&n, "n", argc, argv);
	chCommandLineGet<int>(&n, "size", argc, argv);
	n = (n!=-1)? n:DEFAULT_N;

	// thread per block
	int blockSize = -1;
	chCommandLineGet<int>(&blockSize, "b", argc, argv);
	chCommandLineGet<int>(&blockSize, "blockSize", argc, argv);
	blockSize = (blockSize!=-1)? blockSize:DEFAULT_BLOCKSIZE;

	// memory allocation
	std::cout << "Memory allocation...\t" << std::flush;
		// cpu
	int* cpu_v = (int*)malloc(n*sizeof(int));
	int* out = (int*)malloc(n*sizeof(int));

		// gpu
	int na, nb;
	na = floor(n/2); 	nb = ceil(n/2);

	int *A, *B, *C;
	hipMalloc(&A, na*sizeof(int));
	hipMalloc(&B, nb*sizeof(int));
	hipMalloc(&C, n*sizeof(int));

	if(!A || !B || !C ) {
		std::cout << "memory alloc error" << std::endl;
		return -1;
	}

	std::cout << "done" << std::endl;
	
	// initialization
	std::cout << "Initialization and H2D...\t" << std::flush;
	
	//init_array(cpu_v, n);
	init_array(cpu_v, na, 0);		init_array(cpu_v+na, nb, 0);
	bubbleSort(cpu_v, na);			bubbleSort(cpu_v+na, nb);
	hipMemcpy(A, cpu_v, na*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B, cpu_v+na, nb*sizeof(int), hipMemcpyHostToDevice);
	
	if(n<70) {
		std::cout << std::endl;
		print_array(cpu_v, na);
		print_array(cpu_v+na, nb);
	}
	std::cout << "done" << std::endl << std::endl;

	// Kernel call
	std::cout << "Partitionning...\t" << std::flush;

	ChTimer kernel;
	kernel.start();
	partition2<<<1, blockSize>>>(A, na, B, nb, C);
	hipDeviceSynchronize();
	//msWrapper(cpu_v, n, out, 5);
	kernel.stop();
	std::cout << "done" << std::endl;

	//print_array(out, n);

	// D2H
	std::cout << "transfert D2H...\t" << std::flush;
	hipMemcpy(out, C, n*sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "done" << std::endl;
	
	// compare results
	ChTimer cpuTimer;
	//string filename;
	bool compare_cpu = (chCommandLineGetBool("c", argc, argv))?
		true:chCommandLineGetBool("compare-cpu", argc, argv);
	//bool store = (chCommandLineGetBool("r", argc, argv))?
	//	true:chCommandLineGetBool("store-results", argc, argv);
	if(compare_cpu) {
		std::cout << "cpu sort...\t" << std::flush;
		cpuTimer.start();
		bubbleSort(cpu_v, n);
		cpuTimer.stop();
		std::cout << "done" << std::endl << std::endl;
	}

	// afficher
	std::cout << "Results...\t" << std::flush;
	if(n<70)	print_array(out, n);
	else		std::cout << std::endl;
	std::cout << "\tsorted=" << is_sorted(out, n) << std::endl;
	std::cout << "\tgpu time: " << 1e3*kernel.getTime() << "ms" << std::endl;
	if(compare_cpu)
		std::cout <<"\tcpu time: "<<1e3*cpuTimer.getTime()<<"ms"<<std::endl;

	//free
	free(cpu_v);	free(out);
	hipFree(A);	hipFree(B);
	hipFree(C);

	return 0;
}

void print_help( char* argv) {

	std::cout << "Help:" << std::endl
		<< "  Usage: " << std::endl
		<< "  " << argv << " [options][-n <size>][-b <blockSize>]" << std::endl
		<< std::endl
		<< "  -n|--size" << std::endl
		<< "      size of input array" << std::endl
		<< "  -b|--blockSize" << std::endl
		<< "      size of thread block, only one block is used" << std::endl
		<< std::endl;

}
