#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <chTimer.hpp>
#include <chCommandLine.h>
#include <MergeSort.h>

const static int DEFAULT_N = 12;
const static int DEFAULT_BLOCKSIZE = 4;

// function prototypes
template <typename type> void init_array(type* array, int n, int mod=0);
template <typename type> void cpu_sort(type* array, int n);
template <typename type> void print_array(type* array, int n);
template <typename type> bool is_sorted(type* array, int n);

void print_help( char* argv);

//main
int main(int argc, char* argv[]){
	
	// print help
	bool      help = chCommandLineGetBool("h", argc, argv);
	if(!help) help = chCommandLineGetBool("help", argc, argv);
	if(help)  {
		print_help(argv[0]); return 0;
	}

	// size
	int n = -1;
	chCommandLineGet<int>(&n, "n", argc, argv);
	chCommandLineGet<int>(&n, "size", argc, argv);
	n = (n!=-1)? n:DEFAULT_N;

	// thread per block
	int blockSize = -1;
	chCommandLineGet<int>(&blockSize, "b", argc, argv);
	chCommandLineGet<int>(&blockSize, "blockSize", argc, argv);
	blockSize = (blockSize!=-1)? blockSize:DEFAULT_BLOCKSIZE;

	// memory allocation
	std::cout << "Memory allocation...\t" << std::flush;
		// cpu
	int* cpu_v = (int*)malloc(n*sizeof(int));
	int* out = (int*)malloc(n*sizeof(int));

		// gpu
	int na, nb;
	na = floor(n/2); 	nb = ceil(n/2);

	int *A, *B, *C;
	hipMalloc(&A, na*sizeof(int));
	hipMalloc(&B, nb*sizeof(int));
	hipMalloc(&C, n*sizeof(int));

	if(!A || !B || !C ) {
		std::cout << "memory alloc error" << std::endl;
		return -1;
	}

	std::cout << "done" << std::endl;

	// initialization
	std::cout << "Initialization...\t" << std::flush;

	//init_array(cpu_v, n);
	//cpu_sort(cpu_v, na);		//cpu_sort(cpu_v+na, nb);
	init_array(cpu_v, na, 1);		init_array(cpu_v+na, nb, 1);
	hipMemcpy(A, cpu_v, na*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(B, cpu_v+na, nb*sizeof(int), hipMemcpyHostToDevice);

	if(n<60) {
		std::cout << std::endl;
		print_array(cpu_v, na);
		print_array(cpu_v+na, nb);
	}
	std::cout << "done" << std::endl << std::endl;

	// Kernel call
	std::cout << "Partitionning...\t" << std::flush;

	ChTimer kernel;
	kernel.start();
	partitionning<int><<<1, blockSize>>>(A, na, B, nb, C);
	kernel.stop();
	hipDeviceSynchronize();
	std::cout << "done" << std::endl;

	// D2H
	std::cout << "transfert D2H...\t" << std::flush;
	hipMemcpy(out, C, n*sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "done" << std::endl;
	
	// compare results
	ChTimer cpuTimer;
	//string filename;
	bool compare_cpu = (chCommandLineGetBool("c", argc, argv))?
		true:chCommandLineGetBool("compare-cpu", argc, argv);
	//bool store = (chCommandLineGetBool("r", argc, argv))?
	//	true:chCommandLineGetBool("store-results", argc, argv);
	if(compare_cpu) {
		std::cout << "cpu sort...\t" << std::flush;
		cpuTimer.start();
		cpu_sort(cpu_v, n);
		cpuTimer.stop();
		std::cout << "done" << std::endl << std::endl;
	}

	// afficher
	std::cout << "Results...\t" << std::flush;
	if(n<60)	print_array(out, n);
	else		std::cout << std::endl;
	std::cout << "\tsorted=" << is_sorted(out, n) << std::endl;
	std::cout << "\tgpu time: " << 1e3*kernel.getTime() << "ms" << std::endl;
	if(compare_cpu)
		std::cout <<"\tcpu time: "<<1e3*cpuTimer.getTime()<<"ms"<<std::endl;

	//free
	free(cpu_v);	free(out);
	hipFree(A);	hipFree(B);
	hipFree(C);

	// return 0
	return 0;
}

template <typename type>
void init_array(type* array, int n, int mod) {

	switch(mod) {
	case 0:
		for(int i=0;i<n;i++){
			array[i]=rand()%10;
		}
		break;

	case 1:
		for(int i=0;i<n;i++){
                        array[i]=i;
                }
                break;
	}
}

template <typename type>
void cpu_sort(type* array, int n) {
	bool swap=true;
	for(int i=0;(i<n)&&swap;i++){
		swap = false;
		for(int j=0;(j<n-i-1);j++){
			if(array[j]>array[j+1]){
				int tmp = array[j];
				array[j] = array[j+1];
				array[j+1] = tmp;
				swap = true;
			}
		}
	}
}

template <typename type>
bool is_sorted(type* array, int n) {
	for(int i=0; i<n-1; i++) {
		if(array[i]>array[i+1]) return false;
	}
	return true;
}


template <typename type>
void print_array(type* array, int n) {
	for(int i=0; i<n; i++) {
		std::cout << array[i] << " ";
	}
	std::cout << std::endl;
}

void print_help( char* argv) {

	std::cout << "Help:" << std::endl
		<< "  Usage: " << std::endl
		<< "  " << argv << " [options][-n <size>][-b <blockSize>]" << std::endl
		<< std::endl
		<< "  -n|--size" << std::endl
		<< "      size of input array" << std::endl
		<< "  -b|--blockSize" << std::endl
		<< "      size of thread block, only one block is used" << std::endl
		<< std::endl;

}
