#include "hip/hip_runtime.h"
#include <merge.h>

__host__ __device__ void merge(int* A, int na, int aid, int* B, int nb, int bid,
				int* C, int cid, int load) {

	for(int t=0; t<load; t++) {
		if(aid<na && bid<nb) { // this should always be true...
			if(A[aid] < B[bid]) {
				C[cid+t] = A[aid];	aid++;
			}
			else {
				C[cid+t] = B[bid];	bid++;
			}
		}
		else if(aid<na) {
			 C[cid+t] = A[aid];      aid++;
		}
		else if(bid<nb) {
			C[cid+t] = B[bid];      bid++;
		}
	}
	
}


__global__ void merge2(int* A, int na, int aid, int* B, int nb, int bid,
				int* C, int cid, int load) {

	//int tid = blockIdx.x*blockDim.x+threadIdx.x;	// thread ID
	int index = cid+threadIdx.x;			// starting index in C
	//int tid = threadIdx.x;
	int a, b, offset;
	
	while( index < (cid+load) ) {
		// find path
		if(index==cid) {
			// thread 0 always starts at (0,0)
			a = aid;	b = bid;
		}
		else {
			// search zone: indices of the top-right cell of the central diagonal
			int a_top = aid+threadIdx.x;	// col index (in A)
			int b_top = bid;		// row index (in B)
			
			//int a_top = (aid+threadIdx.x<na)? aid+threadIdx.x:na-1;
			//int b_top = (aid+threadIdx.x<na)? bid:bid+aid+threadIdx.x-na+1;
			
			//int a_top = (aid+threadIdx.x>na)? na:aid+threadIdx.x;
			//int b_top = (aid+threadIdx.x>na)? bid+aid+threadIdx.x-na:bid;
			
			int a_bot = b_top;		// top left col index
			
			if(index==10 || index==11) {
				printf("[%d] (%d,%d) - %d\n", index, a_top, b_top, a_bot);
			}
			
			if(a_top>na) {
					a = na-1;	b = bid+(a_top-na);
			}
			else {
				int cpt=0;
				while(cpt<10000) {	// binary search (dichotomy)
			
					// get mid cell of the (sub-)diagonal
					offset = (a_top - a_bot) / 2;
					a = a_top - offset;		b = b_top + offset;
				
					if(index==10 || index==11) {
						//printf("[%d] (%d,%d)\n", index, a, b);
					}

					if(A[a]>B[b-1]){
						if(A[a-1]<=B[b]){
							break;	// point found
						}
						else{
							// restrict search to lower half
							a_top = a-1;	b_top = b+1;
						}
					}
					else{
						// restrict search to upper half
						a_bot = a+1;
					}
					cpt++;
				}
			}
		}
		// put the element in C
		//if(index==10 || index==11) printf("[%d] (%d,%d)\n", index, a, b);
		if(A[a] < B[b]) {
			C[index] = A[a];
		}
		else {
			C[index] = B[b];
		}
		index+=blockDim.x;
	}
				
}
