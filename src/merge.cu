#include "hip/hip_runtime.h"
#include <merge.h>

__host__ __device__ void merge(int* A, int na, int aid, int* B, int nb, int bid,
				int* C, int cid, int load) {

	for(int t=0; t<load; t++) {
		if(aid<na && bid<nb) { // this should always be true...
			if(A[aid] < B[bid]) {
				C[cid+t] = A[aid];	aid++;
			}
			else {
				C[cid+t] = B[bid];	bid++;
			}
		}
		else if(aid<na) {
			 C[cid+t] = A[aid];      aid++;
		}
		else if(bid<nb) {
			C[cid+t] = B[bid];      bid++;
		}
	}
	
}


__global__ void merge2(int* A, int na, int aid, int* B, int nb, int bid,
				int* C, int cid, int load) {

	if(blockIdx.x==0 && threadIdx.x==0) printf("\t\tMerge %d x %d\n", gridDim.x, blockDim.x);

	int tid = threadIdx.x;	// thread ID
	int index = cid+tid;	// starting index in C
	int a, b, offset;
	
	while( index < (cid+load) ) {	// batch loop

		// find path
		if(index==cid) { // thread 0 always starts at (0,0)
			a = aid;	b = bid;
		}
		if(index==na+nb-1) {	// if this is the last element of the array
			a = na-1;	b = nb-1;
		}
		else {	// binary search
			// search zone:
			int a_top = (aid+tid>na)? na:aid+tid;	// col index (in A)
			int b_top = (aid+tid>na)? index-na:bid;	// row index (in B)
			int a_bot = b_top;	// top left col index
			
			while(true) {
		
				// get mid cell of the (sub-)diagonal
				offset = (a_top - a_bot) / 2;
				a = a_top - offset;		b = b_top + offset;

				if(A[a]>B[b-1]){
					if(A[a-1]<=B[b]){
						break;	// point found
					}
					else{ // restrict search to lower half
						a_top = a-1;	b_top = b+1;
					}
				}
				else{ // restrict search to upper half
					a_bot = a+1;
				}
			}
		}

		// put the element in C
		if(A[a] < B[b]) {
			C[index] = A[a];
		}
		else {
			C[index] = B[b];
		}
		tid+=blockDim.x;
		index+=blockDim.x;
	}
}				
