#include "hip/hip_runtime.h"
#include <merge.h>

__host__ __device__ void merge(int* A, int na, int aid, int* B, int nb, int bid,
				int* C, int cid, int load) {

	for(int t=0; t<load; t++) {
		if(aid<na && bid<nb) { // this should always be true...
			if(A[aid] < B[bid]) {
				C[cid+t] = A[aid];	aid++;
			}
			else {
				C[cid+t] = B[bid];	bid++;
			}
		}
		else if(aid<na) {
			 C[cid+t] = A[aid];      aid++;
		}
		else if(bid<nb) {
			C[cid+t] = B[bid];      bid++;
		}
	}
	
}


__global__ void merge2(int* A, int na, int aid, int* B, int nb, int bid,
				int* C, int cid, int load) {

	if(blockIdx.x==0 && threadIdx.x==0) printf("\t\tMerge %d x %d\n", gridDim.x, blockDim.x);

	int tid = threadIdx.x;	// thread ID
	int index = cid+tid;	// starting index in C
	int a, b, offset;
	
	while( index < (cid+load) ) {	// batch loop

		// find path
		if(index==cid) { // thread 0 always starts at (0,0)
			a = aid;	b = bid;
		}
		else if(index==na+nb-1) {	// if this is the last element of the array
			a = na-1;	b = nb-1;
			C[index] = (A[a]<B[b])? B[b]:A[a];
		}
		else {	// binary search
			// search zone:
			int a_top = (aid+tid>na)? na:aid+tid;	// col index (in A)
			int b_top = (aid+tid>na)? index-na:bid;	// row index (in B)
			int a_bot = ((a_top-aid)>(nb-b_top))? na+b_top-nb:aid;	// top left col index
			
			printf("\t\t[%d] (%d,%d) %d\n", index, a_top, b_top, a_bot);
			
			if(a_top==na && a_bot == na-1) {
				a = na - 1;	b = b_top+1;
			}
			else if(b_top==nb-1) {
				a = a_top;	b = b_top;
			}
			else {
				int cpt=0;
				while(cpt<1000) {
		
					// get mid cell of the (sub-)diagonal
					offset = (a_top - a_bot) / 2;
					a = a_top - offset;		b = b_top + offset;

					if(A[a]>B[b-1]){
						if(A[a-1]<=B[b]){
							break;	// point found
						}
						else{ // restrict search to lower half
							a_top = a-1;	b_top = b+1;
						}
					}
					else{ // restrict search to upper half
						a_bot = a+1;
					}
					cpt++;
				}
			}
		}

		printf("\t\t[%d] (%d,%d)\n", index, a, b);
		// put the element in C
		if(index!=na+nb-1) {
			if(A[a] < B[b]) {
				C[index] = A[a];
			}
			else {
				C[index] = B[b];
			}
		}
		tid+=blockDim.x;
		index+=blockDim.x;
		
	}
}			
