#include "hip/hip_runtime.h"
#include <merge.h>

__host__ __device__ void merge(int* A, int na, int aid, int* B, int nb, int bid,
				int* C, int cid, int load) {

	for(int t=0; t<load; t++) {
		if(aid<na && bid<nb) { // this should always be true...
			if(A[aid] < B[bid]) {
				C[cid+t] = A[aid];	aid++;
			}
			else {
				C[cid+t] = B[bid];	bid++;
			}
		}
		else if(aid<na) {
			 C[cid+t] = A[aid];      aid++;
		}
		else if(bid<nb) {
			C[cid+t] = B[bid];      bid++;
		}
	}
	
}


__global__ void merge2(int* A, int na, int aid, int* B, int nb, int bid,
				int* C, int cid, int load) {

	int tid = threadIdx.x;	// thread ID
	int index = cid+tid;	// starting index in C
	int a, b, offset;
	
	while( index < (cid+load) ) {	// batch loop

		// find path
		if(index==cid) { // thread 0 always starts at (0,0)
			a = aid;	b = bid;
		}
		else {
			// search zone:
			int a_top = aid+tid;	// col index (in A)
			int b_top = bid;	// row index (in B)
			int a_bot = b_top;	// top left col index
			
			if(index==10 || index==11) {
				printf("[%d] (%d,%d) - %d\n", index, a_top, b_top, a_bot);
			}
			
			if(a_top>na) {	// if all elements of A were already taken
					a = na-1;	b = bid+(a_top-na);
			}
			else {
				int cpt=0;
				while(cpt<10000) {	// binary search (dichotomy)
			
					// get mid cell of the (sub-)diagonal
					offset = (a_top - a_bot) / 2;
					a = a_top - offset;		b = b_top + offset;

					if(A[a]>B[b-1]){
						if(A[a-1]<=B[b]){
							break;	// point found
						}
						else{ // restrict search to lower half
							a_top = a-1;	b_top = b+1;
						}
					}
					else{ // restrict search to upper half
						a_bot = a+1;
					}
					cpt++;
				}
			}
		}

		// put the element in C
		if(A[a] < B[b]) {
			C[index] = A[a];
		}
		else {
			C[index] = B[b];
		}
		tid+=blockDim.x;
		index+=blockDim.x;
	}
				
}
